#include "cuda-data-struct.cuh"
#include "cuda-error.cuh"

int CuInitializeData(CudaData* data, std::size_t const& buffer_size)
{
  hipError_t error;
  CudaErrorCheck(error, hipMalloc(&(data->cu_buffer), buffer_size));
  CudaErrorCheck(error, hipMalloc(&(data->cu_mandel_area), sizeof(MandelPlotArea)));
  data->init = true;
  data->cu_buffer_size = buffer_size;
  return 0;
}

int CuUpdateData(CudaData& data, MandelPlotArea const& plot)
{
  hipError_t error;
  CudaErrorCheck(
    error, 
    hipMemcpy(data.cu_mandel_area, &plot, sizeof(MandelPlotArea), hipMemcpyHostToDevice)
  );
  return 0;
}

int CuResizeBuffer(CudaData& data, int const& width, int const& height)
{
  hipError_t error;
  const std::size_t plot_size = (width * 4ull) * height;
  CudaErrorCheck(error, hipFree(data.cu_buffer));
  CudaErrorCheck(error, hipMalloc(&(data.cu_buffer), plot_size));
  data.cu_buffer_size = plot_size;
  return 0;
}
