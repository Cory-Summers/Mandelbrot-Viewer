#include "cuda-error.cuh"
#include <iostream>
constexpr bool FAILURE = 1;
constexpr bool SUCCESS = 0;

bool _CudaErrorCheck(hipError_t const& error)
{
  if (error != hipSuccess)
  {
    std::cerr << hipGetErrorName(error) << "> " << hipGetErrorString(error) << '\n';
    return FAILURE;
  }
  return SUCCESS;
}